#include "hip/hip_runtime.h"
/*
* ARQUITECTURA DE COMPUTADORES
* 2º Curso - Grado en Ingeniería Informática
* Curso 2016/17
*
* PRACTICA 07: "Memoria compartida dinámica"
* >> Utilizar la memoria compartida de la GPU de manera dinámica para calcular
pi mediante una nueva fórmula
*
* AUTOR: - GARCÍA MEDIAVILLA Marina - GARRIDO LABRADOR José Luis
* FECHA: 25/10/2016
*/
///////////////////////////////////////////////////////////////////////////
// includes
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <math.h>
///////////////////////////////////////////////////////////////////////////
// defines
#define BLOQUE 1
#define HILOS 512
///////////////////////////////////////////////////////////////////////////
// DEVICE: función llamada desde el device y ejecutada en el device
__device__ float calcularArea(float inicio, float final, float base){
  float medio = (inicio+final)/2;
  float altura = 4/(1+(medio)*(medio));
  return base*altura;
}
// GLOBAL: función llamada desde el host y ejecutada en el device (kernel)
__global__ void calcularPi(float *pi,int *precition){
  int hilosLanzados = *precition;
  int myID = threadIdx.x; //La posición global en el vector
  float inicio; //Punto inicial en X del rectángulo
  float final; //Punto final en X del rectángulo
  extern __shared__ float area[]; //Area del rectángulo
  float superVar = 1/(float)hilosLanzados; //Contiene la base de cada
  rectángulo
  //Calculamos el área
  inicio = myID * superVar;
  final = (myID + 1) * superVar;
  area[myID] = calcularArea(inicio,final,superVar);
  __syncthreads();
  //Reducción paralela
  if(hilosLanzados%2 != 0){
    if(myID == (hilosLanzados-1)){
      array[0] += array[myID];
    }
  }
  __syncthreads();
  int salto = hilosLanzados/2;
  // Realizamos log2(N) iteraciones
  while(salto)
  {
    // Solo trabajan la mitad de los hilos
    if(myID < salto)
    {
      area[myID] = area[myID] + area[myID+salto];
    }
    __syncthreads();
    if(salto % 2 != 0 && salto!=1){
      if(myID == salto-1){
        array[0] = array[0] + array[myID];
      }
    }
    __syncthreads();
    salto = salto/2;
  }
  __syncthreads();
  // El hilo no.'0' tiene el valor para calcular Pi
  if(myID==0)
  {
    *pi = area[myID];
  }
}
// HOST: función llamada desde el host y ejecutada en el host
/*
* Nombre: clean_stdin
* Descripción: borra el buffer de teclado
* Retorna 1 cuando termina
*/
__host__ int clean_stdin(void) {
  while (getchar() != '\n');
  return 1;
}
///////////////////////////////////////////////////////////////////////////
// MAIN: rutina principal ejecutada en el host
int main(int argc, char** argv) {
  float *dev_pi, *hst_pi;
  int *dev_precition;
  hipSetDevice(0); //Elegimos la tarjeta 1º
  hipEvent_t start,stop; //marcas de eventos
  int precition;
  char c;
  char linea[] =
  "---------------------------------------------------------------------";
  hipDeviceProp_t features; //Propiedades de la tarjeta
  hipGetDeviceProperties(&features, 0); //Obtenemos los datos de la
  tarjeta
  //Pedimos los datos
  do{
    printf("Seleccione la precisión con la que calcular Pi, como
    máximo %d: ",features.maxThreadsPerBlock);
    //
    if (scanf("%d%c", &precition, &c) != 2 || c != '\n') {
      printf("Valor no valido\n");
      clean_stdin();
    }
  } while (precition < 0 || precition > features.maxThreadsPerBlock);
  //Reservamos memoria
  hst_pi = (float *) malloc(sizeof(float));
  hipMalloc((void**)&dev_pi,sizeof(float));
  hipMalloc((void**)&dev_precition,sizeof(int));
  hipMemcpy(dev_precition,&precition,sizeof(int),hipMemcpyHostToDevice);
  //Creamos los eventos
  hipEventCreate(&start);
  hipEventCreate(&stop);
  //Inicializamos el inicio
  hipEventRecord(start,0);
  //Llamamos a la función kernel para calcular pi
  calcularPi <<<BLOQUE,precition,precition*sizeof(float)>>> (dev_pi,
  dev_precition);
  hipMemcpy(hst_pi,dev_pi,sizeof(float),hipMemcpyDeviceToHost);
  //Inicializamos el final
  hipEventRecord(stop,0);
  //Sincronizamos host y device
  hipEventSynchronize(stop);
  //Calculamos el tiempo entre marcas
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime,start,stop);
  //Imprimimos los resultados
  printf("\n%s\nLa memoria compartida disponible
  %dKiB:\n",linea,features.sharedMemPerBlock/1024);
  printf("Los hilos disponibles son %d de los cuales se utilizaron
  %d\n",features.maxThreadsPerBlock,precition);
  printf("El valor de pi calculado es: %f\n",*hst_pi);
  printf("La ejecución se ha realizado sobre %f
  milisegundos\n",elapsedTime);
  printf("\npulsa INTRO para finalizar...");
  getchar();
  return 0;
}
///////////////////////////////////////////////////////////////////////////